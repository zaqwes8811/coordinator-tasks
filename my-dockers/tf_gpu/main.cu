#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "bitset"

#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include <stdio.h>

#include <stdint.h>

using namespace std;
using namespace cv;

typedef unsigned char uchar;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
		true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}

__device__
inline int clamp(int x, int min_, int max_) {
	if (x < min_)
		x = min_;
	else if (x > max_)
		x = max_;
	return x;
}

__device__ int clamp_2d_position(int w, int h, int gx, int gy) {
	if (gx >= w) {
		return -1;
	}
	if (gy >= h)
		return -1;

	int pos = gy * w + gx;
	if (pos >= w * h) {
		return -1;
	}
	return pos;
}

__global__
void cencus_i3_xsobel5(const uchar* in, const uchar* in_xsobel, uint32_t* out,
		int w, int h) {
	int gx = blockIdx.x * blockDim.x + threadIdx.x;
	int gy = blockIdx.y * blockDim.y + threadIdx.y;

	int pos = clamp_2d_position(w, h, gx, gy);
	if (pos < 0) {
		return;
	}

	uint32_t true_res = 0;
	{
		int k1 = 3; //5;//3;
		int hk1 = k1 >> 1;

		uint32_t res = 0;
		uint32_t mask = 0x01 << (k1 * k1 - 1);

		uint32_t Imean = 0;
		for (int dx = -hk1; dx <= hk1; ++dx) {
			for (int dy = -hk1; dy <= hk1; ++dy) {
				int x = clamp(gx + dx, 0, w);
				int y = clamp(gy + dy, 0, h);
				uchar Ie = in[y * w + x];
				Imean += Ie;
			}
		}
		Imean /= k1 * k1;

		uchar I = Imean; //in[pos];
		for (int dx = -hk1; dx <= hk1; ++dx) {
			for (int dy = -hk1; dy <= hk1; ++dy) {
				if (dy == 0 && dx == 0) {
					continue;
				}
				int x = clamp(gx + dx, 0, w);
				int y = clamp(gy + dy, 0, h);

				uchar Ie = in[y * w + x];
				if (I < Ie) {
					res += 1; //mask;
				} else {
					//res &= ~(mask);
				}
				//mask >>= 1;
				res <<= 1;
			}
		}
		res >>= 1;
		true_res = res;
	}

	{
		int k1 = 5; //3;
		int hk1 = k1 >> 1;

		uint32_t res = 0;
		uint32_t mask = 0x01 << (k1 * k1 - 1);

		uint32_t Imean = 0;
		for (int dx = -hk1; dx <= hk1; ++dx) {
			for (int dy = -hk1; dy <= hk1; ++dy) {
				int x = clamp(gx + dx, 0, w);
				int y = clamp(gy + dy, 0, h);
				uchar Ie = in_xsobel[y * w + x];
				Imean += Ie;
			}
		}
		Imean /= k1 * k1;

		uchar I = Imean; //in_xsobel[pos];
		for (int dx = -hk1; dx <= hk1; ++dx) {
			for (int dy = -hk1; dy <= hk1; ++dy) {
				if (dy == 0 && dx == 0) {
					continue;
				}
				int x = clamp(gx + dx, 0, w);
				int y = clamp(gy + dy, 0, h);

				uchar Ie = in_xsobel[y * w + x];
				if (I < Ie) {
					res |= mask;
				} else {
					res &= ~(mask);
				}
				mask >>= 1;
			}
		}
		res >>= 1;
		res <<= 8; // * 3;
		true_res = true_res | res;
	}

	out[pos] = true_res;
}

// [0, 32]
#define MAX_DISP 32
#define WS 7

#define SHIFT 4
#define FILTERED ((-1) << SHIFT)

__global__ void sbm_census(uint32_t* i0, uint32_t* i1, int w, int h,
		short* d_disp, int ws) {
	int gx_0 = blockIdx.x * blockDim.x + threadIdx.x;
	int gy_0 = blockIdx.y * blockDim.y + threadIdx.y;

	int pos_0 = clamp_2d_position(w, h, gx_0, gy_0);
	if (pos_0 < 0) {
		return;
	}

	d_disp[pos_0] = FILTERED;

	// fixme: можно кстати искать на меньшем диапазоне, а не выкидывать
	if (gx_0 < MAX_DISP) {
		return;
	}

	short shd[MAX_DISP];
	int hbs = ws >> 1;

	// SAD -> SHD (Sum of Hamming Distance (SHD))
	// WTA strategy

	// fixme: preload window - may be bad idia

	// (-max, 0]
	for (int d = 0; d < MAX_DISP; ++d) {
		int gx_1 = gx_0 - d;
		int gy_1 = gy_0;

		int shd_acc = 0;

		for (int x = -hbs; x < hbs; ++x) {
			for (int y = -hbs; y < hbs; ++y) {
				int ax0 = clamp(gx_0 + x, 0, w);
				int ay0 = clamp(gy_0 + y, 0, h);

				int ax1 = clamp(gx_1 + x, 0, w);
				int ay1 = clamp(gy_1 + y, 0, h);

				uint32_t I0 = i0[ay0 * w + ax0];
				uint32_t I1 = i1[ay1 * w + ax1];

				uint32_t dI = I0 ^ I1;

				// slow
				shd_acc += __popc(dI);
//				shd_acc += I0 + I1;

			}
		}

		shd[d] = shd_acc;
	}

	// minarg
	float minval = 1e6;
	int arg = -1;
	for (int d = 0; d < MAX_DISP; ++d) {
		if (minval > shd[d]) {
			arg = d;
			minval = shd[d];
		}
	}

	if (arg == -1) {
		d_disp[pos_0] = FILTERED;
		return;
	}

	// subpixel

	// uniqueless

	// text. thr.

	// store
	d_disp[pos_0] = arg << SHIFT;
}

void census(uchar* d_img, uchar* d_img_xsobel, uint32_t* d_img_census, int w,
		int h) {

	const int bs_ = 32;
	const dim3 wg(w / bs_ + 1, h / bs_ + 1);
	const dim3 bs(bs_, bs_);
	// measure
	// https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// Perform SAXPY on 1M elements

	cencus_i3_xsobel5<<<wg, bs>>>(d_img, d_img_xsobel, d_img_census, w, h);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "elapsed:" << milliseconds / 1e3 << endl;
}

struct gpu_matching_handle_t {

	gpu_matching_handle_t(int N, uchar* h_i0, uchar* h_i1, uchar* h_i0_xsobel,
			uchar* h_i1_xsobel) {
		d_i0 = 0;
		d_i0_census = 0;
		d_i1 = 0;
		d_i1_census = 0;

		// im0
		gpuErrchk(hipMalloc(&d_i0, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i0_xsobel, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i0_census, N * sizeof(uint32_t)));

		gpuErrchk(
				hipMemcpy(d_i0, h_i0, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		gpuErrchk(
				hipMemcpy(d_i0_xsobel, h_i0_xsobel, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		// im1
		gpuErrchk(hipMalloc(&d_i1, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i1_xsobel, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i1_census, N * sizeof(uint32_t)));

		gpuErrchk(
				hipMemcpy(d_i1, h_i1, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		gpuErrchk(
				hipMemcpy(d_i1_xsobel, h_i1_xsobel, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		// disp
		gpuErrchk(hipMalloc(&d_disp_i16, N * sizeof(short)));
	}

	~gpu_matching_handle_t() {
		gpuErrchk(hipFree(d_i0));
		gpuErrchk(hipFree(d_i0_xsobel));
		gpuErrchk(hipFree(d_i0_census));
		gpuErrchk(hipFree(d_i1));
		gpuErrchk(hipFree(d_i1_xsobel));
		gpuErrchk(hipFree(d_i1_census));
		gpuErrchk(hipFree(d_disp_i16));
	}

	uchar *d_i0;
	uchar *d_i0_xsobel;
	uint32_t *d_i0_census;
	uchar *d_i1;
	uchar *d_i1_xsobel;
	uint32_t *d_i1_census;

	short* d_disp_i16;
};

int main(void) {

	 string root = "/mnt/d1/datasets/2011_09_26/2011_09_26_drive_0052_sync/";
//	string root = "/mnt/d1/datasets/2011_09_26/2011_09_26_drive_0018_sync/";
//	 string root = "/mnt/d1/datasets/2011_09_26/2011_09_26_drive_0056_sync/";
//	string root = "/mnt/d1/datasets/2011_09_26/2011_09_26_drive_0056_sync/";

	string fi0 = root + "image_00/data/0000000077.png";
	string fi1 = root + "image_01/data/0000000077.png";
	Mat im0 = imread(fi0.c_str(), 0);
	Mat im1 = imread(fi1.c_str(), 0);

	Mat dst0, dst1, xsob0, xsob1;
	double f = 0.25;
	cv::resize(im0, dst0, Size(0, 0), f, f);
	im0 = dst0;
	cv::resize(im1, dst1, Size(0, 0), f, f);
	im1 = dst1;

	//
	int scale = 1;
	int delta = 128;
	int ddepth = CV_8U;
	Scharr(im0, xsob0, ddepth, 0, 1, scale, delta, BORDER_DEFAULT);
	Scharr(im1, xsob1, ddepth, 0, 1, scale, delta, BORDER_DEFAULT);
	// Sobel( im0, xsob0, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
	// Sobel( im1, xsob1, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
	imwrite("out0_sobel.png", xsob0);
	imwrite("out1_sobel.png", xsob1);

	//

	int w = im0.cols;
	int h = im0.rows;
	int N = w * h;

	gpu_matching_handle_t mhandle(N, im0.data, im1.data, xsob0.data,
			xsob1.data);

	//
	//
	// census
	census(mhandle.d_i0, mhandle.d_i0_xsobel, mhandle.d_i0_census, w, h);
	census(mhandle.d_i1, mhandle.d_i1_xsobel, mhandle.d_i1_census, w, h);

	vector<uint32_t> h_i0_census(N);
	vector<uint32_t> h_i1_census(N);

	gpuErrchk(
			hipMemcpy(&h_i0_census[0], mhandle.d_i0_census,
					N * sizeof(uint32_t), hipMemcpyDeviceToHost));
	gpuErrchk(
			hipMemcpy(&h_i1_census[0], mhandle.d_i1_census,
					N * sizeof(uint32_t), hipMemcpyDeviceToHost));

	//
	//
	//
	//
	// https://stackoverflow.com/questions/14581806/can-not-use-cv-32uc1
	Mat A = Mat(h, w, CV_32S, &h_i0_census[0]);

	Mat B;
	A.convertTo(B, CV_8U);
	imwrite("h0_census.png", B);

	for (int i = 0; i < h_i1_census.size(); ++i) {
		// cout << std::bitset<32>(h_i0_census[i]);
		// cout << " " << std::bitset<32>(h_i1_census[i]) << endl;
	}

	A = Mat(h, w, CV_32S, &h_i1_census[0]);
	A.convertTo(B, CV_8U);
	imwrite("h1_census.png", B);

	// matching
	{
		const int tmp = 32;
		vector<short> h_disp(N, 0);
		const dim3 wg(w / tmp + 1, h / tmp + 1);
		const dim3 bs(tmp, tmp);
		// measure
		// https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		sbm_census<<<wg, bs>>>(mhandle.d_i0_census, mhandle.d_i1_census, w, h,
				mhandle.d_disp_i16, WS);

		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		hipEventRecord(stop);

		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		cout << "elapsed:" << milliseconds / 1e3 << endl;

		// store
		gpuErrchk(
				hipMemcpy(&h_disp[0], mhandle.d_disp_i16, N * sizeof(short),
						hipMemcpyDeviceToHost));

		A = Mat(h, w, CV_16S, &h_disp[0]);

		A.convertTo(B, CV_8U);

		cv::resize(B, dst1, Size(0, 0), 1 / f, 1 / f);
		B = dst1;

		imwrite("d_disp.png", B);
	}

}

